#include "hip/hip_runtime.h"
#include "wb.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>


#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif


__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	printf("AM DOING STUFF");

	if (i < len)
	{
		out[i] = in1[i] + in2[i];
	}
}

int main(int argc, char **argv) {
    wbArg_t args;
    int inputLength;
    float *hostInput1;
    float *hostInput2;
	float *hostOutput;
    args = wbArg_read(argc, argv);
    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 =
    (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 =
    (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *)malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");
    wbLog(TRACE, "The input length is ", inputLength);
    wbTime_start(GPU, "Allocating GPU memory.");
	
	float * memInput1 = nullptr;
	hipMalloc(&memInput1, inputLength * sizeof(float));

	float * memInput2 = nullptr;
	hipMalloc(&memInput2, inputLength * sizeof(float));

	float * memOutput = nullptr;
	hipMalloc(&memOutput, inputLength * sizeof(float));

    wbTime_stop(GPU, "Allocating GPU memory.");
    wbTime_start(GPU, "Copying input memory to the GPU.");

	hipMemcpy(memInput1, hostInput1, inputLength * sizeof(float), hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(memInput2, hostInput2, inputLength * sizeof(float), hipMemcpyKind::hipMemcpyHostToDevice);
	//hipMemset(memOutput, 0, inputLength * sizeof(float));

	//@@ Copy memory to the GPU here
    wbTime_stop(GPU, "Copying input memory to the GPU.");
    //@@ Initialize the grid and block dimensions here
	
	int blockSize(256);
	int gridSize((inputLength-1)/blockSize + 1);

    wbTime_start(Compute, "Performing CUDA computation");
	


	vecAdd KERNEL_ARGS2(gridSize, blockSize) (memInput1, memInput2, memOutput, inputLength);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    wbTime_start(Copy, "Copying output memory to the CPU");

	hipMemcpy(hostOutput, memOutput, inputLength * sizeof(float), hipMemcpyKind::hipMemcpyDeviceToHost);
    //@@ Copy the GPU memory back to the CPU here
    wbTime_stop(Copy, "Copying output memory to the CPU");
    wbTime_start(GPU, "Freeing GPU Memory");
	hipFree(memInput1);
	hipFree(memInput2);
	hipFree(memOutput);
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, inputLength);
    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

	getchar();

    return 0;
}
